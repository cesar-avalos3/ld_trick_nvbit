#include <hip/hip_runtime.h>



void FuncGetParam(hipFunction_t func, size_t paramIndex, size_t* paramOffset, size_t* paramSize)
{
	cuFuncGetParamInfo(func, paramIndex, paramOffset, paramSize);
	return;
}
